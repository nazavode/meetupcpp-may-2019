#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstddef>
#include <vector>

__global__ void add_kernel(const int* a, const int* b, int* result, int count) {
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < count) {
        c[id] = a[id] + b[id];
    }
}

std::vector<int> add(const std::vector<int>& a, const std::vector<int>& b) {
    // We are going to operate on the common indexes subset
    std::vector<int> result(std::min(std::size(a), std::size(b)));
    if (std::size(result) == 0) return {};
    const auto byte_size = std::size(result) * sizeof(int);

    // Add some proper RAII to cuda resources
    const auto make_unique_device =
        [](std::size_t size) -> std::unique_ptr<int[], decltype(hipFree)> {
        void* ptr;
        hipMalloc(&ptr, size);
        return {ptr, hipFree};
    };

    // Device allocation
    auto device_a = make_device_buffer(byte_size);
    auto device_b = make_device_buffer(byte_size);
    auto device_result = make_device_buffer(byte_size);

    // Copy host vectors to device
    hipMemcpy(device_a.get(), std::data(host_a), byte_size, hipMemcpyHostToDevice);
    hipMemcpy(device_b.get(), std::data(host_b), byte_size, hipMemcpyHostToDevice);

    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 1024;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n / blockSize);

    // Execute the kernel
    add_kernel<<<gridSize, blockSize>>>(device_a.get(), device_b.get(),
                                        device_result.get(), count);

    // Copy result back to host
    hipMemcpy(std::data(host_result), device_result.get(), byte_size,
               hipMemcpyDeviceToHost);

    return result;
}

std::vector<int> make_dataset(std::size_t size) {
    std::vector<int> dataset(size);
    std::generate(std::begin(dataset), std::end(dataset), [&]() {
        static int value = 0;
        return value++;
    });
    std::random_shuffle(std::begin(dataset), std::end(dataset));
    return dataset;
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cout << "Usage: " << argv[0] << " <items count>" << std::endl;
        return 1;
    }
    const auto count = std::stoll(argv[1]);

    const auto a = make_dataset(count);
    const auto b = make_dataset(count);
    const auto result = add(a, b);
    // Correctness check
    auto expected = std::vector<int>(count);
    std::transform(std::begin(a), std::end(a), std::begin(b), std::begin(expected),
                   std::plus<>{});
    auto [result_it, expected_it] =
        std::mismatch(std::begin(result), std::end(result), std::begin(expected),
                      [](auto a, auto b) { return a == b; });
    if (result_it != std::end(result)) {
        const auto diff_idx = std::distance(std::begin(result), result_it);
        std::cerr << "diff at index " << diff_idx << ": " << *result_it
                  << " != " << *expected_it << std::endl;
        return 1;
    }
}