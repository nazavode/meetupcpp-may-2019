
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cstddef>
#include <vector>
#include <iterator>
#include <string>
#include <iostream>

__global__ void add_kernel(const int* a, const int* b, int* result, int count) {
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check for out of bounds threads
    if (id < count) {
        result[id] = a[id] + b[id];
    }
}

std::vector<int> add(const std::vector<int>& a, const std::vector<int>& b) {
    // We are going to operate on the common indexes subset
    std::vector<int> result(std::min(a.size(), b.size()));
    if (result.size() == 0) return {};

    const auto byte_size = result.size() * sizeof(int);

    int* device_a = nullptr;
    int* device_b = nullptr;
    int* device_result = nullptr;

    // Device allocation
    hipMalloc(&device_a, byte_size);
    hipMalloc(&device_b, byte_size);
    hipMalloc(&device_result, byte_size);

    // Copy host vectors to device
    hipMemcpy(device_a, a.data(), byte_size, hipMemcpyHostToDevice);
    hipMemcpy(device_b, b.data(), byte_size, hipMemcpyHostToDevice);

    // Number of threads in each thread block
    const auto blockSize = 1024;

    // Number of thread blocks in grid
    const auto gridSize = static_cast<int>(
        std::ceil(static_cast<float>(result.size()) / blockSize)
    );

    // Execute the kernel
    add_kernel<<<gridSize, blockSize>>>(device_a, device_b,
                                        device_result, result.size());

    // Copy result back to host
    hipMemcpy(result.data(), device_result, result.size() * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_result);

    return result;
}

std::vector<int> make_dataset(std::size_t size) {
    std::vector<int> dataset(size);
    std::generate(std::begin(dataset), std::end(dataset), [&]() {
        static int value = 0;
        return value++;
    });
    std::random_shuffle(std::begin(dataset), std::end(dataset));
    return dataset;
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cout << "Usage: " << argv[0] << " <items count>" << std::endl;
        return 1;
    }
    const auto count = std::stoll(argv[1]);

    const auto a = make_dataset(count);
    const auto b = make_dataset(count);
    const auto result = add(a, b);
    // Correctness check
    auto expected = std::vector<int>(count);
    std::transform(std::begin(a), std::end(a), std::begin(b), std::begin(expected),
                   std::plus<int>{});
    const auto diff =
        std::mismatch(std::begin(result), std::end(result), std::begin(expected),
                      [](auto a, auto b) { return a == b; });
    const auto result_it = diff.first;
    const auto expected_it = diff.second;
    if (result_it != std::end(result)) {
        const auto diff_idx = std::distance(std::begin(result), result_it);
        std::cerr << "diff at index " << diff_idx << ": " << *result_it
                  << " != " << *expected_it << std::endl;
        return 1;
    }
}
